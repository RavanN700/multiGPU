#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
 
// __global__ void delay(int *dest, int destDevice, int *src, int srcDevice,int num_elems,unsigned long long timeout_clocks = 10000) {                        
// // Wait until the application notifies us that it has completed queuing up the
// // experiment, or timeout and exit, allowing the application to make progress
// long long int start_clock, sample_clock;
// start_clock = clock64();
// hipMemcpyPeer(dest, destDevice, src, srcDevice, sizeof(int) * num_elems);

// while (1) {
// sample_clock = clock64();

// if (sample_clock - start_clock > timeout_clocks) {
// break;
// }
// }
// }
 
 
int main(int argc, char **argv) {

    using namespace std;
    int numGPUs;

    int src;
    int det;
    int memsize;

    hipGetDeviceCount(&numGPUs); // get number of GPUs

    printf("Please enter the source GPU: ");
    scanf("%d", &src);
    printf("\n");
    printf("Please enter the detination GPU: "); 
    scanf("%d", &det);
    printf("\n");
    printf("Please enter the Number of Elements: ") ;
    scanf("%d", &memsize);
    printf("\n");


    // Initilize buffer for src and det GPU
    vector<int *> buffers(numGPUs);
    // Src GPU
    hipSetDevice(src);
    hipMalloc(&buffers[src], memsize * sizeof(int));
    hipMemset(buffers[src], src, memsize * sizeof(int)); // Set buffer[src] to value src
    // Det GPU
    hipSetDevice(det);
    hipMalloc(&buffers[det], memsize * sizeof(int));
    hipMemset(buffers[det], det, memsize * sizeof(int)); // Set buffer[det] to value det


    // Start profiler // nvprof --profile-from-start off
    hipProfilerStart(); 

    // Copy data from src GPU to det GPU
    hipSetDevice(src);
    // delay<<<128, 128>>>(buffers[det], det, buffers[src], src, sizeof(int) * memsize, 10000);
    hipMemcpyPeer(buffers[det], det, buffers[src], src, sizeof(int) * memsize);
    

    // Stop profiler
    hipProfilerStop(); 

    double mb = memsize * sizeof(int) / (double)1e6;
    printf("Size of data transfer (MB): %f\n", mb);


    exit(EXIT_SUCCESS);
 }
 