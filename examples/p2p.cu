#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
 
// Idea of p2p.cu: copy mem from src gpu to det gpu
// Two cases:
// Case 1: using hipMemcpyPeer(d_B, det, d_A, src, size) to copy d_A to d_B. d_A on src GPU, d_B on det GPU
// Case 2: using vecAdd kernel to copy d_A to d_B. d_A on src GPU, d_B, d_C on det GPU

 // Vector kernel 
__global__ void
vecAdd(const int *A, const int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

// initilize vector to be all "1"
static void
initVec(int *vec, int n, int value)
{
    for (int i = 0; i < n; i++)
        vec[i] = value;
}

 
int main(int argc, char **argv) {

    using namespace std;
    // int numGPUs;

    int src=0;
    int det=1;
    int memsize;

    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    struct timeval t1, t2;



    // hipGetDeviceCount(&numGPUs); // get number of GPUs

    // printf("Please enter the source GPU: ");
    // scanf("%d", &src);
    // printf("\n");
    // printf("Please enter the detination GPU: "); 
    // scanf("%d", &det);
    // printf("\n");
    // printf("Please enter the Number of Elements: ") ;
    scanf("%d", &memsize);
    // printf("\n");
    
    // src = 0;
    // det = 1;
    // memsize = 1;


    size_t size = memsize * sizeof(int);


    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize input vectors
    initVec(h_A, memsize, 1);
    initVec(h_B, memsize, 2);
    initVec(h_C, memsize, 100);

    // Src GPU contains vec_A 
    hipSetDevice(src);
    hipMalloc((void**)&d_A, size);  

    // Copy vector A from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // Det GPU contains vec_B and vec_C
    hipSetDevice(det);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);


    // Copy vector B from host memory to device memory
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Make sure src GPU have access to det GPU
    hipSetDevice(src);
    hipDeviceEnablePeerAccess(det, 0);  

    
    int threadsPerBlock = 256;
    int blocksPerGrid = (memsize + threadsPerBlock - 1) / threadsPerBlock;
    
    // Start record time
    // hipEventRecord(start);
    gettimeofday(&t1, 0);    
    

    // Start profiler // nvprof --profile-from-start off
    hipProfilerStart(); 
    

    

    // vecadd kernel
    vecAdd <<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, memsize);

    // Peer to peer memory copy from device src to device det
    // hipMemcpyPeer(d_B, det, d_A, src, size);


    
    // Stop profiler
    hipProfilerStop(); 

    // Stop time record
    // hipEventRecord(stop);
    gettimeofday(&t2, 0);

    // hipEventSynchronize(stop);
    double milliseconds = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;


    // Copy back to host memory in src GPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost); // needed for kernel 
    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost); // test peer2peer memcpy


    double mb = memsize * sizeof(int) / (double)1e6;
    printf("Size of data transfer (MB): %f\n", mb);
    printf("Vector V_A (original value = 1): %d\n",h_A[memsize-1]);
    printf("Vector V_B (original value = 2): %d\n",h_B[memsize-1]);
    printf("Vector V_C (original value = 100): %d\n", h_C[memsize-1]);
    printf("Time (ms): %f\n", milliseconds);
    printf("Bandwith (MB/s): %f\n",mb*1e3/milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    exit(EXIT_SUCCESS);
 }
 