#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
 
 // Vector kernel 
__global__ void
vecAdd(const int *A, const int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

// initilize vector to be all "1"
static void
initVec(int *vec, int n, int value)
{
    for (int i = 0; i < n; i++)
        vec[i] = value;
}

 
int main(int argc, char **argv) {

    using namespace std;
    // int numGPUs;

    int src;
    int det;
    int memsize;

    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    struct timeval t1, t2;



    // hipGetDeviceCount(&numGPUs); // get number of GPUs

    printf("Please enter the source GPU: ");
    scanf("%d", &src);
    printf("\n");
    printf("Please enter the detination GPU: "); 
    scanf("%d", &det);
    printf("\n");
    printf("Please enter the Number of Elements: ") ;
    scanf("%d", &memsize);
    printf("\n");

    size_t size = memsize * sizeof(int);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize input vectors
    initVec(h_A, memsize, 1);
    initVec(h_B, memsize, 2);
    initVec(h_C, memsize, 3);

    // Src GPU contains vec_A and vec_C
    hipSetDevice(src);
    hipMalloc((void**)&d_A, size);  
    hipMalloc((void**)&d_C, size);

    // Copy vector A from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // Det GPU contains vec_B
    hipSetDevice(det);
    hipMalloc((void**)&d_B, size);

    // Copy vector B from host memory to device memory
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Make src and det device both valid
    // int deviceList[4] = {0,1,2,3};
    // hipSetValidDevices(deviceList, 2);
    // hipSetDevice(src);
    int threadsPerBlock = 256;
    int blocksPerGrid = (memsize + threadsPerBlock - 1) / threadsPerBlock;
    
    // Start record time
    // hipEventRecord(start);
    gettimeofday(&t1, 0);    
    

    // Start profiler // nvprof --profile-from-start off
    hipProfilerStart(); 
    

    

    // vecadd kernel
    // vecAdd <<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, memsize);

    // Peer to peer memory copy from device src to device det
    hipSetDevice(src);
    hipMemcpyPeer(d_B, det, d_A, src, size);
    hipSetDevice(det);
    hipMemcpyPeer(d_B, det, d_A, src, size);

    
    // Stop profiler
    hipProfilerStop(); 

    // Stop time record
    // hipEventRecord(stop);
    gettimeofday(&t2, 0);

    // hipEventSynchronize(stop);
    // double milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    double milliseconds = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;


    // Copy back to host memory in src GPU
    // hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost); // test peer2peer memcpy


    double mb = memsize * sizeof(int) / (double)1e6;
    printf("Size of data transfer (MB): %f\n", mb);
    printf("Vector V_A (original value = 1): %d\n",h_A[memsize-1]);
    printf("Vector V_B (original value = 2): %d\n",h_B[memsize-1]);
    // printf("Vector V_C[memsize-1] (original value = 3): %d\n", h_C[memsize-1]);
    printf("Time (ms): %f\n", milliseconds);
    printf("Bandwith (MB/s): %f\n",mb*1e3/milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    exit(EXIT_SUCCESS);
 }
 