#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
 
 // Vector kernel 
__global__ void
vecAdd(const int *A, const int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

 
int main(int argc, char **argv) {

    using namespace std;
    // int numGPUs;

    int src;
    int det;
    int memsize;

    // hipGetDeviceCount(&numGPUs); // get number of GPUs

    printf("Please enter the source GPU: ");
    scanf("%d", &src);
    printf("\n");
    printf("Please enter the detination GPU: "); 
    scanf("%d", &det);
    printf("\n");
    printf("Please enter the Number of Elements: ") ;
    scanf("%d", &memsize);
    printf("\n");


    // Initilize buffer for src and det GPU
    vector<int *> buffers(10);
    // Src GPU
    hipSetDevice(src);
    hipMalloc(&buffers[src], memsize * sizeof(int));     // vec A  
    hipMemset(buffers[src], src, memsize * sizeof(int)); // Set buffer[src] to value src

    hipMalloc(&buffers[9], memsize * sizeof(int)); // vec C
    hipMemset(buffers[9], 9, memsize * sizeof(int));

    // Det GPU
    hipSetDevice(det);
    hipMalloc(&buffers[det], memsize * sizeof(int));
    hipMemset(buffers[det], det, memsize * sizeof(int)); // Set buffer[det] to value det

    hipSetDevice(src);
    hipSetDevice(det);
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (memsize + threadsPerBlock - 1) / threadsPerBlock;
    // Start profiler // nvprof --profile-from-start off
    hipProfilerStart(); 
    

    vecAdd <<<256, 256>>>(buffers[src], buffers[det], buffers[9], memsize);

    // Copy data from src GPU to det GPU
    // hipMemcpyPeer(buffers[det], det, buffers[src], src, sizeof(int) * memsize);
    

    // Stop profiler
    hipProfilerStop(); 

    hipFree(buffers[src]);
    hipFree(buffers[det]);
    hipFree(buffers[9]);


    double mb = memsize * sizeof(int) / (double)1e6;
    printf("Size of data transfer (MB): %f\n", mb);


    exit(EXIT_SUCCESS);
 }
 