#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
 
 // Vector kernel 
__global__ void
vecAdd(const int *A, const int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

// initilize vector to be all "1"
static void
initVec(int *vec, int n)
{
    for (int i = 0; i < n; i++)
        vec[i] = 1;
}

 
int main(int argc, char **argv) {

    using namespace std;
    // int numGPUs;

    int src;
    int det;
    int memsize;

    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;


    // hipGetDeviceCount(&numGPUs); // get number of GPUs

    printf("Please enter the source GPU: ");
    scanf("%d", &src);
    printf("\n");
    printf("Please enter the detination GPU: "); 
    scanf("%d", &det);
    printf("\n");
    printf("Please enter the Number of Elements: ") ;
    scanf("%d", &memsize);
    printf("\n");

    size_t size = memsize * sizeof(int);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize input vectors
    initVec(h_A, 1);
    initVec(h_B, 2);
    initVec(h_C, 10);

    // Src GPU contains vec_A and vec_C
    hipSetDevice(src);
    hipMalloc((void**)&d_A, size);  
    hipMalloc((void**)&d_C, size);

    // Copy vector A from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // Det GPU contains vec_B
    hipSetDevice(det);
    hipMalloc((void**)&d_B, size);

    // Copy vector B from host memory to device memory
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Make src and det device both valid
    int deviceList[2] = {0,1};
    hipSetValidDevices(deviceList, 2);
    int threadsPerBlock = 256;
    int blocksPerGrid = (memsize + threadsPerBlock - 1) / threadsPerBlock;
    
    // Start profiler // nvprof --profile-from-start off
    hipProfilerStart(); 
    
    // vecadd kernel
    vecAdd <<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, memsize);

    

    // Stop profiler
    hipProfilerStop(); 


    // Copy back to host memory in src GPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    double mb = memsize * sizeof(int) / (double)1e6;
    printf("Size of data transfer (MB): %f\n", mb);

    printf("Output vector: %d\n", h_C[0]);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    exit(EXIT_SUCCESS);
 }
 