#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
 
__global__ void delay(volatile int *flag,
    unsigned long long timeout_clocks = 1000000000) {                        
// Wait until the application notifies us that it has completed queuing up the
// experiment, or timeout and exit, allowing the application to make progress
long long int start_clock, sample_clock;
start_clock = clock64();

while (!*flag) {
sample_clock = clock64();

if (sample_clock - start_clock > timeout_clocks) {
break;
}
}
}
 
 
int main(int argc, char **argv) {

    using namespace std;
    int numGPUs;

    int src;
    int det;
    int memsize;
    volatile int *flag = NULL;

    hipGetDeviceCount(&numGPUs); // get number of GPUs

    printf("Please enter the source GPU: ");
    scanf("%d", &src);
    printf("\n");
    printf("Please enter the detination GPU: "); 
    scanf("%d", &det);
    printf("\n");
    printf("Please enter the Number of Elements: ") ;
    scanf("%d", &memsize);
    printf("\n");


    // Initilize buffer for src and det GPU
    vector<int *> buffers(numGPUs);
    // vector<hipStream_t> stream(numGPUs);
    // Src GPU
    hipSetDevice(src);
    // hipStreamCreateWithFlags(&stream[src], hipStreamNonBlocking);
    hipMalloc(&buffers[src], memsize * sizeof(int));
    hipMemset(buffers[src], src, memsize * sizeof(int)); // Set buffer[src] to value src
    // Det GPU
    hipSetDevice(det);
    // hipStreamCreateWithFlags(&stream[det], hipStreamNonBlocking);
    hipMalloc(&buffers[det], memsize * sizeof(int));
    hipMemset(buffers[det], det, memsize * sizeof(int)); // Set buffer[det] to value det

    // Start profiler // nvprof --profile-from-start off
    hipProfilerStart(); 

    // Copy data from src GPU to det GPU
    // hipSetDevice(src);
    // hipStreamSynchronize(stream[src]);
    // hipMemcpyPeerAsync(buffers[det], det, buffers[src], src, sizeof(int) * memsize, stream[src]);
    *flag = 0;
    delay<<<128, 128, 0, stream[i]>>>(flag);
    hipMemcpyPeer(buffers[det], det, buffers[src], src, sizeof(int) * memsize);
    // hipStreamSynchronize(stream[src]);
    
    // Stop profiler
    // hipProfilerStop(); 

    double mb = memsize * sizeof(int) / (double)1e6;
    printf("Size of data transfer (MB): %f\n", mb);


    exit(EXIT_SUCCESS);
 }
 